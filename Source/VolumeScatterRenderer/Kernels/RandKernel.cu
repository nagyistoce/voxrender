#include "hip/hip_runtime.h"
/* ===========================================================================

	Project: GPU based Volume Scatter Renderer
    
	Description: Performs initialization of the CUDA RNG states

    Copyright (C) 2014 Lucas Sherman

	Lucas Sherman, email: LucasASherman@gmail.com

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.

=========================================================================== */

// Include Header
#include "RandKernel.h"

// Include Dependencies
#include "VoxLib/Error/CudaError.h"
#include "VoxLib/Core/Devices.h"

namespace {
namespace filescope {

    // --------------------------------------------------------------------
    //  Generates a set of CRNG states
    // --------------------------------------------------------------------
    __global__ void randKernel(size_t size, hiprandState * state)
    { 	
	    // Establish the image coordinates of this pixel
	    int pos = blockIdx.x * blockDim.x + threadIdx.x;
        if (pos >= size) return;
        hiprand_init(0, pos, 0, &state[pos]);
    }

} // namespace filescope
} // namespace anonymous

namespace vox {

float RandKernel::m_elapsedTime;

// --------------------------------------------------------------------
//  Executes the rand initialization kernel for the active device
// --------------------------------------------------------------------
hiprandState * RandKernel::create(size_t size)
{
    hiprandState * states;
    VOX_CUDA_CHECK(hipMalloc((void**)&states, size*sizeof(hiprandState)));

    // :TODO: Break down large images into multi-pass to reduce overhead global memory per pixel
    
	// Setup the execution configuration
	static const unsigned int BLOCK_SIZE = 16*16;
    dim3 threads(BLOCK_SIZE);
    dim3 blocks((size + threads.x - 1) / threads.x);

    // Batch generate the rand states to avoid timeout
    for (size_t i = 0; i < size; i += 768*768)
    {
        size_t subsize = 768*768;
        if (i + subsize > size)
        {
            subsize = i + subsize - size;
        }

	    // Execute the kernel
        hipEvent_t start, stop;
        VOX_CUDA_CHECK(hipEventCreate(&start));
        VOX_CUDA_CHECK(hipEventRecord(start,0));
        filescope::randKernel<<<blocks,threads>>>(subsize, states+i);
        VOX_CUDA_CHECK(hipDeviceSynchronize());
        VOX_CUDA_CHECK(hipEventCreate(&stop));
        VOX_CUDA_CHECK(hipEventRecord(stop,0));
        VOX_CUDA_CHECK(hipEventSynchronize(stop));

        // Acquire the time for this kernel execution
        VOX_CUDA_CHECK(hipEventElapsedTime(&m_elapsedTime, start, stop));
    }

    return states;
}

// --------------------------------------------------------------------
//  frees a block of global memory holding CRNG state info
// --------------------------------------------------------------------
void RandKernel::destroy(hiprandState * states)
{
    VOX_CUDA_CHECK(hipFree(states));
}

} // namespace vox